#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 33
using namespace std;

const int blocksize = 32;

// __global__ decorator signifies a kernel that can be called from the host
__global__ void vec_con_0(int *a, int *b, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x ;
	for (int i = 0; i < n; i++)
	{

		if (i < 5)
		{
			if (i > 2 )
				continue;
				//break;
			else
			{
				if (id < 16)
				{
					a[id] += 1;
				}
				else
					continue;
			}
		}
		else  
		{
			if (i < 8)
			{
				if (id >15)
				{
					a[id] += 2;
				}
				else
					break;
			}
			else
				break;
		}
		a[id] += 1;
	}
}


__global__ void vec_con_1(int *a, int *b, int n)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < vect_len)
		for (int j = 0; j < n; j++)
		{
			if (id < vect_len / 3)
				continue;
			else
			{
				if (id < vect_len / 3 * 2)
					a[id] +=10;
				else
				{
					b[id] += 4;
					if (b[id] == 200)
						break;
				}
				a[id] += 1;
			}
			b[id] += 1;
		}
}
	
__global__ void vec_con_2(int *a, int *b, int n)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id > 5)
	{	
		int i = 0;
		do
		{
			if (id > 10)
				a[id] += b[id];
			else
			{
				if (id < 7)
					break;
				else
				{
					a[id] += 1;
					continue;
				}
			}
		} while((id + i++) < 20);
	}
}


int main(){

	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result_v1=(int*)malloc(vect_size);
	int * result_v2=(int*)malloc(vect_size);
 	bool flag, flag_1, flag_2;

	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid((vect_len + blocksize - 1)/blocksize, 1 , 1);
	// call device kernel
	//vect_add<<<dimGrid, dimBlock>>>(ad, bd);
	vec_con_0<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	//Verify
	flag = true;

	for(int i = 0; i < vect_len; i++)
	{
		if (i < 16)
		{
			if (result_v1[i] != i + 6)
			{
				cout << " Test 0 Error at " << i << " expecting "
				<< i + 6 << " getting " << result_v1[i] <<endl;
				flag = false;
			}
			
		}
		else
		{
			if (result_v1[i] != i + 9)
			{
				cout << "Test 0 Error at " << i << " expecting "
				<< i + 9 << " getting " << result_v1[i] <<endl;
				flag = false;
			}
		}

		
	}

	if(flag)
		cout << "Verification test 0 passes." <<endl;

	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );

	vec_con_1<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	flag_1 = true;

	for (int id = 0; id < vect_len; id++)
	{
		int a = id;
		int b = 2 * id;

		for (int j = 0; j < 10; j++)
		{
			if (id < vect_len / 3)
				continue;
			else
			{
				if (id < vect_len /3 * 2)
					a +=10;
				else
				{
					b += 4;
					if (b == 200)
						break;
				}
				a += 1;
			}
			b += 1;
		}

		if (a != result_v1[id])
		{
			cout << "Test 1 Error at a " << id << " expecting "
				<< a << " getting " << result_v1[id] <<endl;
			flag_1 = false;
		}

		if (b != result_v2[id])
		{
			cout << "Test 1 Error at b " << id << " expecting "
				<< b << " getting " << result_v2[id] <<endl;
			flag_1 = false;
		}
			
	}

	if(flag_1)
		cout << "Verification test 1 passes." <<endl;



	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );

	vec_con_2<<<dimGrid, dimBlock>>>(ad, bd, 10);
	hipMemcpy( result_v1, ad, vect_size, hipMemcpyDeviceToHost );
	hipMemcpy( result_v2, bd, vect_size, hipMemcpyDeviceToHost );

	int* a = (int*)calloc(vect_len, sizeof(int));
	int* b = (int*)calloc(vect_len, sizeof(int));

	for (int i = 0; i < vect_len; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	for (int id = 0; id < vect_len; id++)
	{
		if (id > 5)
		{	
			int i = 0;
			do
			{
				if (id > 10)
					a[id] += b[id];
				else
				{
					if (id < 7)
						break;
					else
					{
						a[id] += 1;
						continue;
					}
				}
			} while((id + i++) < 20);
		}
	}

	flag_2 = true;

	for(int i = 0; i < vect_len; i++)
	{
		if (result_v1[i] != a[i])
		{
			cout << "Test2 failed at a " << i
				<< " expecting " << a[i] 
				<< " getting " << result_v1[i]<< endl;
			flag_2 = false;
		}
		if (result_v2[i] != b[i])
		{
			cout << "Test2 failed at b " << i
				<< " expecting " << b[i]
				<< " getting " << result_v2[i]<< endl;
			flag_2 = false;
		}
	}

	if(flag_2)
		cout << "Verification test 2 passes." <<endl;

	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result_v1);
	free(result_v2);
	return EXIT_SUCCESS;
}



